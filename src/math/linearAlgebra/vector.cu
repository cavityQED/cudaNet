#include "vector.h"



template <typename dataType>
void Vector<dataType>::resize(const shape_t& shape)
{
	m_shape = shape;
	m_shape.pad();
	hipMalloc((void**)&m_buffer, m_shape.span() * sizeof(dataType));

}













template class Vector<double>;